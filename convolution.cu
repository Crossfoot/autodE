
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define HEIGHT 333

/* Pseudocode for convolution given a 3D array (x, y, pixel) (to be done not on GPU since I don't know how to allocate and copy 2D memory yet)
convolution(baseArray, filter, targetArr)  
  direction = [[-1, -1], [-1, 0], [-1, 1]
               [0, -1], [0, 0], [0, 1]
               [1, -1], [1, 0], [1, 1]]
  for i from [1, 334]: # iterate across rows
    for j from [1, 501]: # each pixel in each row
      pixel = [0, 0, 0]
      for k from [0, 8]: # Filter iterating
        for l from [0, 2]:  # RGB in each pixel
          pixel[l] += baseArray[i + direction[k][0]][j + direction[k][1]][l] * filter[k]



*/

__global__ void convolution(int **a, float *b, int **c) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  /* This function is going to be designed with 0 padding in mind */
  float pixel[] = {0, 0, 0};
  int direction[] = {-HEIGHT - 1, -HEIGHT, -HEIGHT + 1, 
                        -1,          0,         1,
                       HEIGHT - 1, HEIGHT, HEIGHT + 1};
  for (int i = 0; i < 9; i++) {
    for (int j = 0; j < 3; j++) {
      int target_idx = idx + direction[i];
      if (target_idx )
      pixel[j] += a[idx + direction[i]][j] * b[i]; /* this is best case scenario, no out of bounds */
    }
  }
  int intPixel[3];
  /*for (int i = 0; i < 3; i++) {
    intPixel[i] = (int)pixel[i];
    printf("%d ", intPixel[i]);
  }
  printf("\n");*/
  c[idx] = intPixel;
}

/* Pseudocode for readPPM 

  Open file for reading
  read the first 3 lines, (color type, picture size [500x333], and how many colors are supported [255])
  store the picture size in an array [500, 333]
  store empty line [[0, 0, 0] ... [0, 0, 0]]
  For i to 333:
    push [0, 0, 0]
    for j to 500:
      push pixel
    push [0, 0, 0]
  push empty line [[0, 0, 0] ... [0, 0, 0]]
  return array of pixels with 0 padding

*/

int **readPPM(char *filename) {
  FILE *file = fopen(filename, "r");
  char *buffer = (char *)malloc(sizeof(char) * 2048), *token;
  size_t size = 2048;
  int xy[2], i = 0, j, **ppmArr, *pixel;
  /* process first 3 lines (color type, pic size, RGB size) */
  getline(&buffer, &size, file); /* p3 */ 
  getline(&buffer, &size, file); /* size */
  token = strtok(buffer, " ");
  do {
    xy[i++] = atoi(token);
  } while (token = strtok(NULL, " "));
  ppmArr = (int**)malloc(sizeof(int*) * xy[0] * xy[1]);
  getline(&buffer, &size, file);
  i = 0;
  while (getline(&buffer, &size, file) > 0) {
    pixel = (int*)malloc(sizeof(int) * 3);  
    pixel[0] = atoi(strtok(buffer, " "));
    for (j=1; j < 3; j++) {
      pixel[j] = atoi(strtok(NULL, " "));
    }
    ppmArr[i] = pixel;
    i++;
  }
  fclose(file);
  return ppmArr; 
}

void readPPMArr(int **ppmarr) {
  for (int i = 0; i < 500 * 333; i++) {
    printf("[");
    for (int k = 0; k < 3; k++) {
      printf("%d, ", ppmarr[i][k]);
    }
    printf("]\n");
  }
}

int main(int argc, char *argv[]) {
  int **ga, **gc, **ppm, **result=NULL;
  float *gb;
  float filter[] = {0, 0, 0,
                  0, 0, 1,
                  0, 0, 0};
  ppm = readPPM(argv[1]);
  printf("sizeof ppm = %d\n", ppm[0][0]);
  /* readPPMArr(ppm); */
  /* PPM read correctly */
  /* Remember, cudaMALLOC needs a pointer to give the memory address to */
  hipMalloc((void ***) &ga, sizeof(int) * 500 * 333); /* input */
  hipMalloc((void **) &gb, sizeof(float) * 3 * 3); /* filter */
  hipMalloc((void ***) &gc, sizeof(int) * 500 * 333); /* output */
  hipMemcpy(ga, ppm, sizeof(int) * 500 * 333, hipMemcpyHostToDevice);
  hipMemcpy(gb, filter, sizeof(float) * 3 * 3, hipMemcpyHostToDevice);
  convolution<<<500,333>>>(ga,gb,gc);
  hipMemcpy(result, gc, sizeof(int) * 500 * 333, hipMemcpyDeviceToHost);
  printf("To here\n");
  readPPMArr(result);
  hipFree(ga);
  hipFree(gb);
  hipFree(gc);
  return 0;
}
